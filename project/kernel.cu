#include <hip/hip_runtime.h>

__global__ void histograma_kernel(int* d_arr, int* d_res, int n, int grupos, int min, int intervalo) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int indice = (d_arr[tid] - min) / intervalo;

        if (indice < grupos) {
            atomicAdd(&d_res[indice], 1);
        } else {
            atomicAdd(&d_res[indice - 1], 1);
        }
    }
}

extern "C" void invoke_histograma_cuda(int* d_arr, int* d_res, int n, int grupos, int min, int intervalo) {
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    histograma_kernel<<<grid_size, block_size>>>(d_arr, d_res, n, grupos, min, intervalo);
}